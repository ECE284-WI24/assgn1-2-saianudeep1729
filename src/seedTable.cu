#include "hip/hip_runtime.h"
#include "seedTable.cuh"
#include <stdio.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

/**
 * Prints information for each available GPU device on stdout
 */
void printGpuProperties () {
    int nDevices;

    // Store the number of available GPU device in nDevicess
    hipError_t err = hipGetDeviceCount(&nDevices);

    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipGetDeviceCount failed!\n");
        exit(1);
    }

    // For each GPU device found, print the information (memory, bandwidth etc.)
    // about the device
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Device memory: %lu\n", prop.totalGlobalMem);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}

/**
 * Allocates arrays on the GPU device for (i) storing the compressed sequence
 * (ii) kmer offsets of the seed table (iii) kmer positions of the seed table
 * Size of the arrays depends on the input sequence length and kmer size
 */
void GpuSeedTable::DeviceArrays::allocateDeviceArrays (uint32_t* compressedSeq, uint32_t seqLen, uint32_t kmerSize) {
    hipError_t err;

    d_seqLen = seqLen;
    uint32_t compressedSeqLen = (seqLen+15)/16;
    uint32_t maxKmers = (uint32_t) pow(4,kmerSize)+1;

    // Only (1)allocate and (2)transfer the 2-bit compressed sequence to GPU.
    // This reduces the memory transfer and storage overheads
    // 1. Allocate memory
    err = hipMalloc(&d_compressedSeq, compressedSeqLen*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    // 2. Transfer compressed sequence
    err = hipMemcpy(d_compressedSeq, compressedSeq, compressedSeqLen*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    // Allocate memory on GPU device for storing the kmer offset array
    err = hipMalloc(&d_kmerOffset, maxKmers*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    // Allocate memory on GPU device for storing the kmer position array
    // Each element is size_t (64-bit) because an intermediate step uses the
    // first 32-bits for kmer value and the last 32-bits for kmer positions
    err = hipMalloc(&d_kmerPos, (seqLen-kmerSize+1)*sizeof(size_t));
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipMalloc failed!\n");
        exit(1);
    }
    hipDeviceSynchronize();
}

/**
 * Free allocated GPU device memory for different arrays
 */
void GpuSeedTable::DeviceArrays::deallocateDeviceArrays () {
    hipFree(d_compressedSeq);
    hipFree(d_kmerOffset);
    hipFree(d_kmerPos);
}

/**
 * Finds kmers for the compressed sequence creates an array with elements
 * containing the 64-bit concatenated value consisting of the kmer value in the
 * first 32 bits and the kmer position in the last 32 bits. The values are
 * stored in the arrary kmerPos, with i-th element corresponding to the i-th
 * kmer in the sequence
 *
 * ASSIGNMENT 2 TASK: parallelize this function
 */
__global__ void kmerPosConcat(
    uint32_t* d_compressedSeq,
    uint32_t d_seqLen,
    uint32_t kmerSize,
    size_t* d_kmerPos) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;

    // HINT: Values below could be useful for parallelizing the code
    //int bs = blockDim.x;
    //int gs = gridDim.x;

    uint32_t N = d_seqLen;
    uint32_t k = kmerSize;

    // Helps mask the non kmer bits from compressed sequence. E.g. for k=2,
    // mask=0x1111 and for k=3, mask=0x111111
    uint32_t mask = (1 << 2*k)-1;
    size_t kmer = 0;

    // HINT: the if statement below ensures only the first thread of the first
    // block does all the computation. This statement might have to be removed
    // during parallelization
    if ((bx == 0) && (tx == 0)) {
        for (uint32_t i = 0; i <= N-k; i++) {
            uint32_t index = i/16;
            uint32_t shift1 = 2*(i%16);
            if (shift1 > 0) {
                uint32_t shift2 = 32-shift1;
                kmer = ((d_compressedSeq[index] >> shift1) | (d_compressedSeq[index+1] << shift2)) & mask;
            } else {
                kmer = d_compressedSeq[index] & mask;
            }

            // Concatenate kmer value (first 32-bits) with its position (last
            // 32-bits)
            size_t kPosConcat = (kmer << 32) + i;
            d_kmerPos[i] = kPosConcat;
        }
    }
}

/**
 * Generates the kmerOffset array using the sorted kmerPos array consisting of
 * the kmer and positions. Requires iterating through the kmerPos array and
 * finding indexes where the kmer values change, depending on which the
 * kmerOffset values are determined.
 *
 * ASSIGNMENT 2 TASK: parallelize this function
 */
__global__ void kmerOffsetFill(
    uint32_t d_seqLen,
    uint32_t kmerSize,
    uint32_t numKmers,
    uint32_t* d_kmerOffset,
    size_t* d_kmerPos) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    // HINT: Values below could be useful for parallelizing the code
    //int bs = blockDim.x;
    //int gs = gridDim.x;

    uint32_t N = d_seqLen;
    uint32_t k = kmerSize;

    size_t mask = ((size_t) 1 << 32)-1;
    uint32_t kmer = 0;
    uint32_t lastKmer = 0;

    // HINT: the if statement below ensures only the first thread of the first
    // block does all the computation. This statement might have to be removed
    // during parallelization
    if ((bx == 0) && (tx == 0)) {
        for (uint32_t i = 0; i <= N-k; i++) {
            kmer = (d_kmerPos[i] >> 32) & mask;
            if (kmer != lastKmer) {
                for (auto j=lastKmer; j<kmer; j++) {
                    d_kmerOffset[j] = i;
                }
            }
            lastKmer = kmer;
        }

        // For all kmers lexicographically larger than the lexicographically
        // largest kmer in the sequence, set offset to N-k
        // HINT: This loop can also be parallelized (e.g. using thread block
        // that encounters position N-k)
        for (auto j=lastKmer; j<numKmers; j++) {
            d_kmerOffset[j] = N-k;
        }
    }
}

/**
 * Masks the first 32 bits of the elements in the kmerPos array
 *
 * ASSIGNMENT 2 TASK: parallelize this function
 */
__global__ void kmerPosMask(
    uint32_t d_seqLen,
    uint32_t kmerSize,
    size_t* d_kmerPos) {

    int tx = threadIdx.x;
    int bx = blockIdx.x;

    // HINT: Values below could be useful for parallelizing the code
    //int bs = blockDim.x;
    //int gs = gridDim.x;

    uint32_t N = d_seqLen;
    uint32_t k = kmerSize;

    size_t mask = ((size_t) 1 << 32)-1;
    if ((bx == 0) && (tx == 0)) {
        for (uint32_t i = 0; i <= N-k; i++) {
            d_kmerPos[i] = d_kmerPos[i] & mask;
        }
    }
}

/**
 * Constructs seed table, consisting of kmerOffset and kmerPos arrrays
 * on the GPU.
*/
void GpuSeedTable::seedTableOnGpu (
    uint32_t* compressedSeq,
    uint32_t seqLen,

    uint32_t kmerSize,

    uint32_t* kmerOffset,
    size_t* kmerPos) {

    // ASSIGNMENT 2 TASK: make sure to appropriately set the values below
    int numBlocks = 1; // i.e. number of thread blocks on the GPU
    int blockSize = 1; // i.e. number of GPU threads per thread block

    kmerPosConcat<<<numBlocks, blockSize>>>(compressedSeq, seqLen, kmerSize, kmerPos);

    // Parallel sort the kmerPos array on the GPU device using the thrust
    // library (https://thrust.github.io/)
    thrust::device_ptr<size_t> kmerPosPtr(kmerPos);
    thrust::sort(kmerPosPtr, kmerPosPtr+seqLen-kmerSize+1);

    uint32_t numKmers = pow(4, kmerSize);;
    kmerOffsetFill<<<numBlocks, blockSize>>>(seqLen, kmerSize, numKmers, kmerOffset, kmerPos);
    kmerPosMask<<<numBlocks, blockSize>>>(seqLen, kmerSize, kmerPos);

    // Wait for all computation on GPU device to finish. Needed to ensure
    // correct runtime profiling results for this function.
    hipDeviceSynchronize();
}

/**
 * Prints the fist N(=numValues) values of kmer offset and position tables to
 * help with the debugging of Assignment 2
 */
void GpuSeedTable::DeviceArrays::printValues(int numValues) {
    uint32_t* kmerOffset = new uint32_t[numValues];
    size_t* kmerPos = new size_t[numValues];

    hipError_t err;

    err = hipMemcpy(kmerOffset, d_kmerOffset, numValues*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(kmerPos, d_kmerPos, numValues*sizeof(size_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    printf("i\tkmerOffset[i]\tkmerPos[i]\n");
    for (int i=0; i<numValues; i++) {
        printf("%i\t%u\t%zu\n", i, kmerOffset[i], kmerPos[i]);
    }
}

